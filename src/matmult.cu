#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#include "matmult.cuh"

Matrix* MatCreateEmpty(int height, int width)
{
    Matrix* X = (Matrix*)malloc(sizeof(Matrix));
    MatInit(X, height, width);
    return X;
}

Matrix* MatCreateHost(int height, int width)
{
    Matrix* X = MatCreateEmpty(height, width);

    X->elements = (float*)malloc(X->size);
    memset(X->elements, 0, X->size);
    X->elements_malloc = X->elements;
    return X;
}

Matrix* MatCreateGPU(int height, int width)
{
    Matrix* X = MatCreateEmpty(height, width);

    hipMalloc(&X->elements, X->size);
    hipMemset(X->elements, 0, X->size);
    X->elements_cudaMalloc = X->elements;
    return X;
}

void MatFree(Matrix** X)
{
    if (!X || !*X)
        return;
    if ((*X)->elements_malloc)
        free((*X)->elements_malloc);
    if ((*X)->elements_cudaMalloc)
        hipFree((*X)->elements_cudaMalloc);
    free(*X);
    *X = NULL;
}

// Matrix multiplication kernel called by MatMultGPU() - basic version
__global__ void MatMult_k0(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    SetElement(&C, r, c, MatMultElement(&A, &B, r, c));
}

// Matrix multiplication kernel called by MatMultGPU() - optimized version
// Should be run this way:
// size_t sharedMemSize = 2 * BLOCK_SIZE * BLOCK_SIZE * sizeof(float); // Total size for As and Bs
// MatMulKernel<<<gridDim, blockDim, sharedMemSize>>>(A, B, C);
__global__ void MatMult_k1(Matrix MatA, Matrix MatB, Matrix MatC)
{
    extern __shared__ char sharedMemory[];

    int R = blockIdx.y;
    int C = blockIdx.x;
    int r = threadIdx.y;
    int c = threadIdx.x;
    int w = blockDim.x;
    int W = MatA.width / w;

    float* Asubs = (float*)sharedMemory;
    float* Bsubs = (float*)&(sharedMemory[w * w * sizeof(float)]);

    Matrix Asub, Bsub, Csub;
    MatInit(&Asub, w, w);
    MatInit(&Bsub, w, w);
    MatInit(&Csub, w, w);

    GetSubMatrix(&MatC, R, C, w, &Csub);
    // Each thread computes one element of Csub
    float Csub_rc = 0;
    // C_{R,C} = \sum_{K=0}^{W-1} A_{R,K} B_{K,C}
    for (int K = 0; K < W; K++) {
        GetSubMatrix(&MatA, R, K, w, &Asub);
        GetSubMatrix(&MatB, K, C, w, &Bsub);
        __syncthreads();
        Asubs[r * w + c] = GetElement(&Asub, r, c);
        Bsubs[r * w + c] = GetElement(&Bsub, r, c);
        __syncthreads();
        // Csub_{r,c} = \sum_{k=0}^{w-1} A_{r,k} B_{k,c}
        for (int k = 0; k < w; k++) {
            Csub_rc += Asubs[r * w + k] * Bsubs[k * w + c];
        }
    }
    SetElement(&Csub, r, c, Csub_rc);
}

void MatMultGPU(const Matrix* A, const Matrix* B, Matrix* C, bool optimized)
{
    assert(A->width == B->height);
    assert(A->height == C->height);
    assert(B->width == C->width);

    // Load A to device memory
    Matrix* d_A = MatCreateGPU(A->height, A->width);
    hipMemcpy(d_A->elements, A->elements, A->size, hipMemcpyHostToDevice);

    // Load B to device memory
    Matrix* d_B = MatCreateGPU(B->height, B->width);
    hipMemcpy(d_B->elements, B->elements, B->size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix* d_C = MatCreateGPU(C->height, C->width);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B->width / dimBlock.x, A->height / dimBlock.y);
    if (optimized) {
        size_t sharedMemSize = 2 * dimBlock.x * dimBlock.y * sizeof(float); // Total size for As and Bs
        MatMult_k1<<<dimGrid, dimBlock, sharedMemSize>>>(*d_A, *d_B, *d_C);
    } else {
        MatMult_k0<<<dimGrid, dimBlock>>>(*d_A, *d_B, *d_C);
    }

    // Read C from device memory
    hipMemcpy(C->elements, d_C->elements, C->size, hipMemcpyDeviceToHost);

    // Free device memory
    MatFree(&d_A);
    MatFree(&d_B);
    MatFree(&d_C);
}

void MatMultHost(const Matrix* A, const Matrix* B, Matrix* C)
{
    assert(A->width == B->height);
    assert(A->height == C->height);
    assert(B->width == C->width);

    for (int r = 0; r < A->height; r++) {
        for (int c = 0; c < B->width; c++) {
            SetElement(C, r, c, MatMultElement(A, B, r, c));
        }
    }
}
void MatPrint(Matrix* A, const char name[])
{
    printf("%s = [\n", name);
    for (int i = 0; i < A->height; i++) {
        int j = 0;
        for (; j < A->width - 1; j++) {
            printf("% 5.1f ", GetElement(A, i, j));
        }
        printf("% 5.1f\n", GetElement(A, i, j));
    }
    printf("]\n");
}

bool MatEqual(Matrix* A, Matrix* B, float tol)
{
    if (A->height != B->height || A->width != B->width)
        return false;
    for (int r = 0; r < A->height; r++)
        for (int c = 0; c < A->width; c++) {
            if (fabs(GetElement(A, r, c) - GetElement(B, r, c)) > tol)
                return false;
        }
    return true;
}
